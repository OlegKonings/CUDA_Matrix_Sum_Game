#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <fstream>
#include <sstream>
#include <utility>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <string>
#include <cmath>
//#include <map>
#include <ctime>
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include "hip/hip_runtime.h"
#include ""
#include <Windows.h>
#include <MMSystem.h>
#pragma comment(lib, "winmm.lib")
#define _CRTDBG_MAP_ALLOC
#include <crtdbg.h>
using namespace std;

typedef long long ll;


#define _DTH hipMemcpyDeviceToHost
#define _DTD hipMemcpyDeviceToDevice
#define _HTD hipMemcpyHostToDevice

#define THREADS 256//do not change
#define MEGA 33232930569601LL//ditto

const int blockSize0=16384;//ditto

long long POW_SEVEN_ARR[17];

bool InitMMTimer(UINT wTimerRes);
void DestroyMMTimer(UINT wTimerRes, bool init);

inline int get_adj_size(const long long num_elem){
	double p=double(num_elem)/double(MEGA);
	if(p>0.8)return 6;
	else if(p>0.6)return 4;
	else if(p>0.4)return 3;
	else if(p>0.2)return 2;
	else
		return 1;
}
inline int get_dynamic_block_size(const int adj_size,const int blkSize){
	return (1<<(adj_size-1))*blkSize;//chk
}

__device__ long long BigPow(long long num, int exp){
	long long ret=1LL;
	while(exp){
		if(exp&1)ret*=num;
		exp>>=1;
		num*=num;
	}
	return ret;
}

void show_arr(const int *Arr, const int d){
	cout<<"board= \n";
	for(int i=0;i<d;i++){
		cout<<Arr[i]<<' ';
		if((i+1)%4==0)cout<<'\n';
	}
	cout<<'\n';
}
void CPU_derive_from_num(long long num, int *Arr, const int digits,const int range, const long long *POw_Arr){
	long long a;
	for(int i=digits-1;i>0;i--){
		a=long long(range-1);
		while(a*POw_Arr[i]>num){a--;}
		Arr[i]=int(a);
		num-=a*POw_Arr[i];
	}
	Arr[0]=int(num);
}


template<int blockWork>
__global__ void GPU_step0(int *best_val, long long *bnum){

	const long long offset=long long(threadIdx.x)+long long(blockIdx.x)*long long(blockWork);
	const int reps=blockWork>>8;
	const int warpIndex = threadIdx.x%32;

	 __shared__ int blk_best[8];
     __shared__ int2 mask_val[8];

	int Arr[16];
	long long pos;//a
	int ii=0,jj,tot=0;
	int2 mask_as_int2,t2;

	for(;ii<reps;ii++){
		pos=offset+long long(ii*THREADS);
		Arr[15]=t2.x=int(pos/4747561509943LL);
		pos-=long long(t2.x)*4747561509943LL;
		Arr[14]=t2.x=int(pos/678223072849LL);
		pos-=long long(t2.x)*678223072849LL;
		Arr[13]=t2.x=int(pos/96889010407LL);
		pos-=long long(t2.x)*96889010407LL;
		Arr[12]=t2.x=int(pos/13841287201LL);
		pos-=long long(t2.x)*13841287201LL;
		Arr[11]=t2.x=int(pos/1977326743LL);
		pos-=long long(t2.x)*1977326743LL;
		t2.y=int(pos);
		Arr[10]=t2.x=t2.y/282475249;
		t2.y-=t2.x*282475249;
		Arr[9]=t2.x=t2.y/40353607;
		t2.y-=t2.x*40353607;
		Arr[8]=t2.x=t2.y/5764801;
		t2.y-=t2.x*5764801;
		Arr[7]=t2.x=t2.y/823543;
		t2.y-=t2.x*823543;
		Arr[6]=t2.x=t2.y/117649;
		t2.y-=t2.x*117649;
		Arr[5]=t2.x=t2.y/16807;
		t2.y-=t2.x*16807;
		Arr[4]=t2.x=t2.y/2401;
		t2.y-=t2.x*2401;
		Arr[3]=t2.x=t2.y/343;
		t2.y-=t2.x*343;
		Arr[2]=t2.x=t2.y/49;
		t2.y-=t2.x*49;
		Arr[1]=t2.x=t2.y/7;
		t2.y-=t2.x*7;	
		Arr[0]=t2.y;

		jj=int(Arr[0]+Arr[1]+Arr[2]+Arr[3]==10)+int(Arr[4]+Arr[5]+Arr[6]+Arr[7]==10)+
			int(Arr[8]+Arr[9]+Arr[10]+Arr[11]==10)+int(Arr[12]+Arr[13]+Arr[14]+Arr[15]==10)+
			int(Arr[0]+Arr[4]+Arr[8]+Arr[12]==10)+int(Arr[1]+Arr[5]+Arr[9]+Arr[13]==10)+
			int(Arr[2]+Arr[6]+Arr[10]+Arr[14]==10)+int(Arr[3]+Arr[7]+Arr[11]+Arr[15]==10)+
			int(Arr[0]+Arr[5]+Arr[10]+Arr[15]==10)+int(Arr[3]+Arr[6]+Arr[9]+Arr[12]==10);

		if(jj>tot){
			tot=jj;
			pos=offset+long long(ii*THREADS);
			mask_as_int2=*reinterpret_cast<int2 *>(&pos);
		}
	}

	for(ii=16;ii>0;ii>>=1){
		jj=__shfl(tot,warpIndex+ii);
		t2.x=__shfl(mask_as_int2.x,warpIndex+ii);
        t2.y=__shfl(mask_as_int2.y,warpIndex+ii);
		if(jj>tot){
			tot=jj;
			mask_as_int2=t2;
		}
	}

	if(warpIndex==0){
		blk_best[threadIdx.x>>5]=tot;
		mask_val[threadIdx.x>>5]=mask_as_int2;
	}
	__syncthreads();

	if(threadIdx.x==0){
		tot=blk_best[0];
		t2=mask_val[0];
		if(blk_best[1]>tot){
			tot=blk_best[1];
			t2=mask_val[1];
		}
		if(blk_best[2]>tot){
			tot=blk_best[2];
			t2=mask_val[2];
		}
		if(blk_best[3]>tot){
			tot=blk_best[3];
			t2=mask_val[3];
		}
		if(blk_best[4]>tot){
			tot=blk_best[4];
			t2=mask_val[4];
		}
		if(blk_best[5]>tot){
			tot=blk_best[5];
			t2=mask_val[5];
		}
		if(blk_best[6]>tot){
			tot=blk_best[6];
			t2=mask_val[6];
		}
		if(blk_best[7]>tot){
			tot=blk_best[7];
			t2=mask_val[7];
		}

		best_val[blockIdx.x]=tot;
		bnum[blockIdx.x]=*reinterpret_cast<long long *>(&t2);
	}
}

__global__ void last_step(int *best_val, long long *bnum,const long long rem_start, const long long bound, const int num_blox){

	const long long offset=long long(threadIdx.x)+rem_start;
	const int warpIndex = threadIdx.x%32;

	__shared__ int blk_best[8];
    __shared__ int2 mask_val[8];

	int Arr[16];
	long long pos,adj=0LL;
	int ii=1,jj,tot=0;
	int2 mask_as_int2,t2;

	for(;(offset+adj)<bound;ii++){
		pos=offset+adj;

		Arr[15]=t2.x=int(pos/4747561509943LL);
		pos-=long long(t2.x)*4747561509943LL;
		Arr[14]=t2.x=int(pos/678223072849LL);
		pos-=long long(t2.x)*678223072849LL;
		Arr[13]=t2.x=int(pos/96889010407LL);
		pos-=long long(t2.x)*96889010407LL;
		Arr[12]=t2.x=int(pos/13841287201LL);
		pos-=long long(t2.x)*13841287201LL;
		Arr[11]=t2.x=int(pos/1977326743LL);
		pos-=long long(t2.x)*1977326743LL;
		t2.y=int(pos);
		Arr[10]=t2.x=t2.y/282475249;
		t2.y-=t2.x*282475249;
		Arr[9]=t2.x=t2.y/40353607;
		t2.y-=t2.x*40353607;
		Arr[8]=t2.x=t2.y/5764801;
		t2.y-=t2.x*5764801;
		Arr[7]=t2.x=t2.y/823543;
		t2.y-=t2.x*823543;
		Arr[6]=t2.x=t2.y/117649;
		t2.y-=t2.x*117649;
		Arr[5]=t2.x=t2.y/16807;
		t2.y-=t2.x*16807;
		Arr[4]=t2.x=t2.y/2401;
		t2.y-=t2.x*2401;
		Arr[3]=t2.x=t2.y/343;
		t2.y-=t2.x*343;
		Arr[2]=t2.x=t2.y/49;
		t2.y-=t2.x*49;
		Arr[1]=t2.x=t2.y/7;
		t2.y-=t2.x*7;	
		Arr[0]=t2.y;

		jj=int(Arr[0]+Arr[1]+Arr[2]+Arr[3]==10)+int(Arr[4]+Arr[5]+Arr[6]+Arr[7]==10)+
			int(Arr[8]+Arr[9]+Arr[10]+Arr[11]==10)+int(Arr[12]+Arr[13]+Arr[14]+Arr[15]==10)+
			int(Arr[0]+Arr[4]+Arr[8]+Arr[12]==10)+int(Arr[1]+Arr[5]+Arr[9]+Arr[13]==10)+
			int(Arr[2]+Arr[6]+Arr[10]+Arr[14]==10)+int(Arr[3]+Arr[7]+Arr[11]+Arr[15]==10)+
			int(Arr[0]+Arr[5]+Arr[10]+Arr[15]==10)+int(Arr[3]+Arr[6]+Arr[9]+Arr[12]==10);

		if(jj>tot){
			tot=jj;
			pos=offset+adj;
			mask_as_int2=*reinterpret_cast<int2 *>(&pos);
		}
		adj=(long long(ii)<<8LL);
	}

	adj=0LL;
	for(ii=1;(threadIdx.x+int(adj))<num_blox;ii++){
		jj=(threadIdx.x+int(adj));
		if(best_val[jj]>tot){
			tot=best_val[jj];
			pos=bnum[jj];
			mask_as_int2=*reinterpret_cast<int2 *>(&pos);
		}
		adj=(long long(ii)<<8LL);
	}

	for(ii=16;ii>0;ii>>=1){
		jj=__shfl(tot,warpIndex+ii);
		t2.x=__shfl(mask_as_int2.x,warpIndex+ii);
        t2.y=__shfl(mask_as_int2.y,warpIndex+ii);
		if(jj>tot){
			tot=jj;
			mask_as_int2=t2;
		}
	}

	if(warpIndex==0){
		blk_best[threadIdx.x>>5]=tot;
		mask_val[threadIdx.x>>5]=mask_as_int2;
	}
	__syncthreads();

	if(threadIdx.x==0){
		tot=blk_best[0];
		t2=mask_val[0];
		if(blk_best[1]>tot){
			tot=blk_best[1];
			t2=mask_val[1];
		}
		if(blk_best[2]>tot){
			tot=blk_best[2];
			t2=mask_val[2];
		}
		if(blk_best[3]>tot){
			tot=blk_best[3];
			t2=mask_val[3];
		}
		if(blk_best[4]>tot){
			tot=blk_best[4];
			t2=mask_val[4];
		}
		if(blk_best[5]>tot){
			tot=blk_best[5];
			t2=mask_val[5];
		}
		if(blk_best[6]>tot){
			tot=blk_best[6];
			t2=mask_val[6];
		}
		if(blk_best[7]>tot){
			tot=blk_best[7];
			t2=mask_val[7];
		}

		best_val[0]=tot;
		bnum[0]=*reinterpret_cast<long long *>(&t2);
	}
}


int main(){


	hipError_t err;

	POW_SEVEN_ARR[0]=1LL;
	for(int i=1;i<=16;i++){
		POW_SEVEN_ARR[i]=7LL*POW_SEVEN_ARR[i-1];
	}
	//long long num=7LL;
	//const int range=7;
	const int digits=16;
	int *Board=(int *)malloc(digits*sizeof(int));
	
	//CPU_derive_from_num(num,Board,digits,range,POW_SEVEN_ARR);
	//show_arr(Board,digits);
	const long long range=POW_SEVEN_ARR[16];
	const int adj_size=get_adj_size(range);
	const int temp_blocks_sz=get_dynamic_block_size(adj_size,blockSize0);
	const int num_blx=int(range/long long(temp_blocks_sz));
	const long long rem_start=range-(range-long long(num_blx)*long long(temp_blocks_sz));
	std::cout<<"\nnum_blx= "<<num_blx<<'\n';

	int GPU_answer=0;
	long long GPU_board=0LL;

	int *best_val;
	long long *bnum;

	err=hipMalloc((void**)&best_val,num_blx*sizeof(int));
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMalloc((void**)&bnum,num_blx*sizeof(long long));
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	
    UINT wTimerRes = 0;
	DWORD GPU_time=0;
    bool init = InitMMTimer(wTimerRes);
    DWORD startTime=timeGetTime();


	GPU_step0<blockSize0><<<num_blx,THREADS>>>(best_val,bnum);
	err = hipDeviceSynchronize();
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	last_step<<<1,THREADS>>>(best_val,bnum,rem_start,range,num_blx);
	err = hipDeviceSynchronize();
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	err=hipMemcpy(&GPU_answer,best_val,sizeof(int),_DTH);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	err=hipMemcpy(&GPU_board,bnum,sizeof(long long),_DTH);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}


	DWORD endTime = timeGetTime();
	GPU_time=endTime-startTime;
	DestroyMMTimer(wTimerRes, init);


	err=hipFree(best_val);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipFree(bnum);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	cout<<"\nGPU timing= "<<GPU_time<<'\n';
	cout<<"\nOptimal score = "<<GPU_answer<<'\n';
	CPU_derive_from_num(GPU_board,Board,digits,7,POW_SEVEN_ARR);
	show_arr(Board,digits);
	cout<<"number = "<<GPU_board<<'\n';


	free(Board);

	err=hipDeviceReset();
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	return 0;
}

bool InitMMTimer(UINT wTimerRes){
	TIMECAPS tc;
	if (timeGetDevCaps(&tc, sizeof(TIMECAPS)) != TIMERR_NOERROR) {return false;}
	wTimerRes = min(max(tc.wPeriodMin, 1), tc.wPeriodMax);
	timeBeginPeriod(wTimerRes); 
	return true;
}

void DestroyMMTimer(UINT wTimerRes, bool init){
	if(init)
		timeEndPeriod(wTimerRes);
}
